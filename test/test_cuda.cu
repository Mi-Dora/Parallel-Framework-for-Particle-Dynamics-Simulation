#include "kernels.h"
#include "device.h"
#include "particle.h"
#include <hip/hip_runtime.h>



int main(int argc, char** argv) {

    const std::string inputFilename = "../data/data10000.txt";
    const double timeStep = 0.1;
    int cell_size = 10;

    chunk_particles_t* chunkParticles = nullptr;
    load_particles(inputFilename, &chunkParticles);


    double *d_pos, *d_vel, *d_acc, *d_feats, *d_timestep;
    uint64_t *d_n_particle, *d_n_dim, *d_n_feat;
    device_allocate_init(&d_pos, &d_vel, &d_acc, &d_feats, &d_n_particle, &d_n_dim, &d_n_feat, &d_timestep, chunkParticles, timeStep);

    dim3 grid_size, block_size;
    size_t shmem_size;
    kernel_params_init(grid_size, block_size, shmem_size, cell_size);
    
    hipEvent_t start, stop;
    cuErrChk(hipEventCreate(&start));
    cuErrChk(hipEventCreate(&stop));

    float cur_time = 0.;
    for(int iter=0; iter<1000; iter++) {
        printf("iter=%d\n", iter);
        cuErrChk(hipEventRecord(start, NULL));
        compute_kernel <<< grid_size, block_size, shmem_size >>> (
                d_pos, d_vel, d_acc, d_feats, d_n_particle, d_n_dim, d_n_feat, d_timestep);
        // kernel(grid_size, block_size, shmem_size, d_pos, d_vel, d_acc, d_feats, d_n_particle, d_n_dim, d_n_feat, d_timestep);
        cuErrChk(hipEventRecord(stop, NULL));
        cuErrChk(hipEventSynchronize(stop));
        cur_time = 0;
        cuErrChk(hipEventElapsedTime(&cur_time, start, stop));

        if(iter%50 == 0) {
            std::string outputFilename = "../data/test_updates/"  + std::to_string(iter) + ".txt";
            store_particles(outputFilename, chunkParticles);
        }
    }

    free_particles(chunkParticles);
    return 0;
}