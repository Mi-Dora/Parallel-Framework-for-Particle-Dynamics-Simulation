#include "hip/hip_runtime.h"
#include "kernels.h"


// __device__
// inline uint64_t 3dto1d(uint64_t x_idx, uint64_t y_idx, uint64_t z_idx){
//     return z_idx * gridDim.x * gridDim.y + y_idx * gridDim.x + x_idx;
// }

__device__
void get_distance_vec(const double* vecA, const double* vecB, const int ndim, double** vecRet) {
    double* ret;
    double* a = const_cast<double*>(vecA);
    double* b = const_cast<double*>(vecB);
    ret = static_cast<double*>(malloc(sizeof(double)*ndim));
    *vecRet = ret;
    for(int n=0; n<ndim; n++) 
        *(ret++) = *(a++) - *(b++);
}

__device__
double get_length(const double* vec, const int ndim) {
    double ret = 0.0;
    double* _vec = const_cast<double*>(vec);
    for(int n=0; n<ndim; n++) {
        ret += (*_vec) * (*_vec);
        _vec++;
    }
    return sqrt(ret);
}

__device__
void d_gravityUpdateAcceleration(particle_t* one, particle_t* another) {
    const double G = 6.67e-11;
    // double* mass1 = one->features;
    double* mass2 = another->features;

    double* positionDiff = nullptr;
    // printf("%d ", one->ndim);
    get_distance_vec(one->position, another->position, one->ndim, &positionDiff);
    double distant = get_length(positionDiff, one->ndim);
    distant = distant * distant * distant;
    distant = G / distant * (*mass2);
    double* acc = one->acceleration;
    double* positionDiff_ = positionDiff;
    for(int n=0; n<one->ndim; n++)
        *(acc++) += *(positionDiff_++) * distant;
    free(positionDiff);
}

__global__ 
void compute_kernel(double *__restrict__ pos, 
                    double *__restrict__ vel,
                    double *__restrict__ acc,
                    double *__restrict__ feats,
                    uint64_t *__restrict__ n_particle,
                    uint64_t *__restrict__ n_dim,
                    uint64_t *__restrict__ n_feat,
                    double *__restrict__ timestep)
{
    uint64_t bid = blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x;
    uint64_t tid = bid * blockDim.x + threadIdx.x;
    uint64_t n_thread = gridDim.x * gridDim.y * gridDim.z * blockDim.x;
    uint64_t _n_dim = *n_dim;
    particle_t *one = static_cast<particle_t*>(malloc(sizeof(particle_t)));
    particle_t *another = static_cast<particle_t*>(malloc(sizeof(particle_t)));
    one->ndim = _n_dim;
    another->ndim = _n_dim;

    for (uint64_t i = tid; i < (*n_particle); i += n_thread){
        one->position = pos+i*_n_dim;
        one->velocity = vel+i*_n_dim;
        one->acceleration = acc+i*_n_dim;
        one->features = feats+i*(*n_feat);
        for (uint64_t j = 0; j < (*n_particle); j++){
            if (i == j) continue;
            another->position = pos+j*_n_dim;
            another->velocity = vel+j*_n_dim;
            another->acceleration = acc+j*_n_dim;
            another->features = feats+j*(*n_feat);
            d_gravityUpdateAcceleration(one, another);
        }
    }
    for (uint64_t i = tid; i < (*n_particle); i += n_thread){
        one->position = pos+i*_n_dim;
        one->velocity = vel+i*_n_dim;
        one->acceleration = acc+i*_n_dim;
        for(int n = 0; n < _n_dim; n++){
            *(one->velocity+n) += *(one->acceleration+n) * (*timestep);
            *(one->position+n) += *(one->velocity+n) * (*timestep);
        }
    }

}

// void kernel(dim3& grid_size, dim3& block_size, size_t& shmem_size, 
//             double* d_pos, double* d_vel, double* d_acc, double* d_feats,
//             uint64_t* d_n_particle, uint64_t* d_n_dim, uint64_t* d_n_feat, double* d_timestep){
//     compute_kernel <<< grid_size, block_size, shmem_size >>> (
//                 d_pos, d_vel, d_acc, d_feats, d_n_particle, d_n_dim, d_n_feat, d_timestep);
// }

// __global__ 
// void scatter_kernel(double const *__restrict__ pos, 
//                     double const *__restrict__ vel,
//                     double const *__restrict__ acc,
//                     double const *__restrict__ feats,
//                     uint64_t const *__restrict__ n_particle,
//                     uint64_t const *__restrict__ n_dim,
//                     uint64_t const *__restrict__ n_feat,
//                     double const *__restrict__ world_size,
//                     uint64_t const *__restrict__ counts
//                     ){
//     uint64_t bid = blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x;
//     uint64_t tid = bid * blockDim.x + threadIdx.x;
//     uint64_t n_thread = gridDim.x * gridDim.y * gridDim.z * blockDim.x;
//     double edge_len = world_size / gridDim.x;
//     // double x_min = blockIdx.x * edge_len;
//     // double y_min = blockIdx.y * edge_len;
//     // double z_min = blockIdx.z * edge_len;
//     // double x_max = x_min + edge_len;
//     // double y_max = y_min + edge_len;
//     // double z_max = z_min + edge_len;
//     uint64_t x_idx, y_idx, z_idx, count;

//     // count for particles in each grid
//     for (int i = tid; i < n_particle; i += n_thread){
//         x_idx = pos[i*n_dim] / edge_len;
//         y_idx = pos[i*n_dim+1] / edge_len;
//         z_idx = pos[i*n_dim+2] / edge_len;
//          atomicAdd(count[3dto1d(x_idx, y_idx, z_idx)], 1);
//     }

//     // 
//     if (blockIdx.x == 0){

//     }



// }

// __global__ 
// void compute_cutoff_kernel(particle_t const *__restrict__ grided_particles,
//         uint64_t const *__restrict__ offset);
